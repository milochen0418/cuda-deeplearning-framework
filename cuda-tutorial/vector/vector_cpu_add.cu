#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

using namespace std;
/*
__global__ void mykernel(void) {
}

int main(void) {
	  mykernel<<<1,1>>>();
	    printf("CPU Hello World!\n");
	      return 0;
}
*/

#define N 10000000

void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add(out, a, b, N);
    /*
    for(int i = 0; i < N; i++){
        printf("[%d] -> %f\n", i, out[i]);
    }
    */
    return 0;
}

